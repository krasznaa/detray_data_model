/** Detray Data Model project, part of the ACTS project (R&D line)
 *
 * (c) 2021 CERN for the benefit of the ACTS project
 *
 * Mozilla Public License Version 2.0
 */

// Local include(s).
#include "detraydm/allocators/managed_allocator_base.hpp"
#include "detraydm/utils/cuda_error_handling.hpp"

// CUDA include(s).
#include <hip/hip_runtime.h>

namespace detraydm::cuda {

   void* managed_allocator_base::cuda_allocate( std::size_t nBytes ) {

      void* ptr = nullptr;
      DETRAYDM_CUDA_ERROR_CHECK( hipMallocManaged( &ptr, nBytes ) );
      return ptr;
   }

   void managed_allocator_base::cuda_deallocate( void* ptr ) {

      DETRAYDM_CUDA_ERROR_CHECK( hipFree( ptr ) );
   }

} // namespace detraydm::cuda
