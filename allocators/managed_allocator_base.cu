// Copyright (C) 2021 Attila Krasznahorkay. All rights reserved.

// Local include(s).
#include "managed_allocator_base.hpp"
#include "core/cuda_error_check.cuh"

// CUDA include(s).
#include <hip/hip_runtime.h>

namespace detray::cuda {

   void* managed_allocator_base::cuda_allocate( std::size_t nBytes ) {

      void* ptr = nullptr;
      DETRAY_CUDA_ERROR_CHECK( hipMallocManaged( &ptr, nBytes ) );
      return ptr;
   }

   void managed_allocator_base::cuda_deallocate( void* ptr ) {

      DETRAY_CUDA_ERROR_CHECK( hipFree( ptr ) );
   }

} // namespace detray::cuda
